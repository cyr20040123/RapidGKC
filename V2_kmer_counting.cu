#include "hip/hip_runtime.h"
// #define TIMER

#define CUDA_CHECK(call) \
if((call) != hipSuccess) { \
    hipError_t err = hipGetLastError(); \
    cerr << "CUDA error calling \""#call"\", code is " << err << ": " << hipGetErrorString(err) << endl; \
    exit(1); \
}
#define CUFILE_STATUS_CHECK(cuerr, lineno) \
if (cuerr.err != CU_FILE_SUCCESS) { \
    cerr << "cuFile error calling line #" << lineno << ", code is " << cuerr.err << endl; \
    exit(1); \
} \
if (cuerr.cu_err != hipSuccess) { \
    hipError_t err = hipGetLastError(); \
    cerr << "cuFile error calling line #" << lineno << ", code is " << cuerr.cu_err <<"|"<< err << ": " << hipGetErrorString(err) << endl; \
    exit(1); \
}

#define NULL_POS 0xFFFFFFFFFFFFFFFFUL

// #include "nvcomp/gdeflate.hpp"
// #include "nvcomp.hpp"

#include <fcntl.h> // open
#include <unistd.h> // close
// #include "cufile.h"

#include <thrust/host_vector.h>
#include <thrust/device_vector.h>
#include <thrust/sort.h>
#include <thrust/sequence.h>
// #include <thrust/scan.h>
#include <thrust/remove.h>
#include <thrust/execution_policy.h>

// #include "types.h"
// #include "V2_superkmers.hpp"
// #include <vector>
#include "kmer_counting.hpp"
#include "utilities.hpp"
#include <fstream>
using namespace std;
// using namespace nvcomp;

extern Logger *logger;

struct sameasprev {
    sameasprev() {}
    __host__ __device__
        bool operator()(const T_kmer& x, const T_kmer& y) const { 
            return x==y;
        }
};
struct canonicalkmer {
    canonicalkmer() {}
    __host__ __device__
        T_kmer operator()(const T_kmer& x, const T_kvalue k) const {
            T_kmer x1 = ~x, res=0;
            for (T_kvalue i=0; i<k; i++) {
                res = (res << 2) | (x1 & 0b11);
                x1 = x1 >> 2;
            }
            return res < x ? res : x;
        }
};
struct replaceidx {
    replaceidx() {}
    __host__ __device__
        T_read_len operator()(const T_read_len& x, const T_read_len& y) const {
            return x*y;
        }
};
struct is_zero {
    __host__ __device__
        bool operator()(const T_read_len x)
        {
            return x==0;
        }
};

__device__ void _process_bytes (size_t beg, size_t end, byte* d_skms, T_kmer *d_kmers, unsigned long long *d_kmer_store_pos, T_kvalue k) {
    // if called, stop until at least one skm is processed whatever end is exceeded
    T_kmer kmer_mask = T_kmer(0xffffffffffffffff>>(64-k*2));
    size_t i;
    T_kmer kmer;
    T_kvalue kmer_bases; // effective bases
    unsigned long long store_pos;
    byte indicator, ii;
    byte beg_selector[4] = {0, 0b00000011, 0b00001111, 0b00111111};
    byte end_selector[4] = {0, 0b00110000, 0b00111100, 0b00111111};
    // Optimization: use ulonglong4 to store 4 kmers at a time
    for (i = beg; i < end; i++) { // i: byte
        // generate the first k-mer
        kmer = 0;
        kmer_bases = 0;
        while (kmer_bases <= k-3) {
            indicator = (d_skms[i]>>6) & 0b11;
            kmer <<= indicator * 2; // why >>5: >>6*2 (2 bits per base)
            kmer |= d_skms[i] & beg_selector[indicator];
            kmer_bases += indicator;
            i++;
        }
        ii = 0; 
        if (kmer_bases < k) { // process the last byte of the first kmer if necessary
            kmer <<= (k-kmer_bases)*2;
            kmer |= (d_skms[i] & end_selector[k-kmer_bases]) >> ((BYTE_BASES-(k-kmer_bases))*2);
            ii = k-kmer_bases; // ii: bases used of the current byte
        }
        store_pos = atomicAdd(d_kmer_store_pos, 1);
        d_kmers[store_pos] = kmer;
        // printf("%llu\n", kmer);
        
        // generate and store the next kmers
        indicator = (d_skms[i]>>6) & 0b11;
        while ((indicator == BYTE_BASES) | (ii < indicator)) { // full block or ii not end
            kmer = ((kmer << 2) | ((d_skms[i] >> ((BYTE_BASES-ii-1)*2)) & 0b11)) & kmer_mask;
            store_pos = atomicAdd(d_kmer_store_pos, 1);
            d_kmers[store_pos] = kmer;
            // printf("%llu\n", kmer);
            ii = (ii+1) % BYTE_BASES;
            i += (ii == 0);
            indicator = (d_skms[i]>>6) & 0b11;
        }
    }
}
__device__ size_t _find_full_nonfull_pos (size_t beg, size_t end, byte* d_skms) {
    byte FN_pos_found = 0; // 0: not found, 1: find full byte, 2: find non-full block after a full
    size_t i;
    for (i = beg; (FN_pos_found<2) & (i < end); i++) {
        FN_pos_found |= ((d_skms[i] & 0b11000000) == 0b11000000); // if full block found, beg_pos_found=1
        FN_pos_found <<= ((d_skms[i] & 0b11000000) < 0b11000000); // if non-full block found, beg_pos_found*=2
    }
    return (FN_pos_found>=2) * i + (FN_pos_found<2) * NULL_POS; // return the next position after a full and nonfull
}
// __global__ void GPU_Extract_Kmers (byte* d_skms, size_t tot_bytes, T_kmer *d_kmers, unsigned long long *d_kmer_store_pos, T_kvalue k) {
//     int n_t = blockDim.x * gridDim.x;
//     int tid = blockDim.x * blockIdx.x + threadIdx.x;
//     size_t bytes_per_thread = (tot_bytes + n_t - 1) / n_t; // min: 1
//     size_t i, search_ending; // which byte to process
//     size_t beg_byte_pos, end_byte_pos;
//     for (i = tid*bytes_per_thread; i/*+bytes_per_thread*/ < tot_bytes; i += n_t*bytes_per_thread) {
//         // printf("i: %llu %llu\n",i,bytes_per_thread);
//         // find begin byte:
//         beg_byte_pos = i==0 ? 0 : _find_full_nonfull_pos(i, i+bytes_per_thread+1, d_skms); // if i==0 begin position is ULL_MAX+1=0, begins from 0
//         // find end byte: (make sure the last full byte is in the area of at least the next thread)
//         search_ending = i+2*bytes_per_thread < tot_bytes ? i+2*bytes_per_thread : tot_bytes;
//         end_byte_pos = _find_full_nonfull_pos (i+bytes_per_thread, search_ending, d_skms);
//         end_byte_pos = (end_byte_pos < search_ending) * end_byte_pos + (end_byte_pos >= search_ending) * search_ending;
//         if (beg_byte_pos < tot_bytes) {
//             // printf("%llu process %llu %llu (%d %llu)\n",tot_bytes, beg_byte_pos, end_byte_pos, tid, i);
//             // printf("%llu %llu\n",beg_byte_pos,end_byte_pos);
//             _process_bytes(beg_byte_pos, end_byte_pos, d_skms, d_kmers, d_kmer_store_pos, k);
//         }
//     }
//     return;
// }

__global__ void GPU_Extract_Kmers (byte* d_skms, size_t tot_bytes, T_kmer *d_kmers, unsigned long long *d_kmer_store_pos, T_kvalue k) {
    int n_t = blockDim.x * gridDim.x;
    int tid = blockDim.x * blockIdx.x + threadIdx.x;
    size_t bytes_per_thread = (tot_bytes + n_t - 1) / n_t; // min: 1
    size_t i, search_ending1, search_ending2; // which byte to process
    size_t beg_byte_pos, end_byte_pos;
    for (i = tid*bytes_per_thread; i/*+bytes_per_thread*/ < tot_bytes; i += n_t*bytes_per_thread) {
        // printf("i: %llu %llu\n",i,bytes_per_thread);
        // find begin byte:
        search_ending1 = i+bytes_per_thread-1 < tot_bytes ? i+bytes_per_thread-1 : tot_bytes;
        beg_byte_pos = i<2 ? 0 : _find_full_nonfull_pos(i-2, search_ending1, d_skms); // if i==0 begin position is ULL_MAX+1=0, begins from 0
        // find end byte: (make sure the last full byte is in the area of at least the next thread)
        search_ending2 = i+2*bytes_per_thread < tot_bytes ? i+2*bytes_per_thread : tot_bytes;
        end_byte_pos = _find_full_nonfull_pos (search_ending1-1, search_ending2, d_skms);
        end_byte_pos = (end_byte_pos < search_ending2) * end_byte_pos + (end_byte_pos >= search_ending2) * search_ending2; // DEBUGGED DONE
        // end_byte_pos = (end_byte_pos < tot_bytes) * end_byte_pos + (end_byte_pos >= tot_bytes) * tot_bytes;
        if (beg_byte_pos < end_byte_pos) {
            // printf("%llu process %llu %llu (%d %llu)\n",tot_bytes, beg_byte_pos, end_byte_pos, tid, i);
            // printf("%llu %llu\n",beg_byte_pos,end_byte_pos);
            _process_bytes(beg_byte_pos, end_byte_pos, d_skms, d_kmers, d_kmer_store_pos, k);
        }
    }
    return;
}

#ifdef DEBUG
__global__ void GPU_Extract_Kmers_test (byte* d_skms, size_t tot_bytes, T_kmer *d_kmers, unsigned long long *d_kmer_store_pos, T_kvalue k) {
    if (blockDim.x * blockIdx.x + threadIdx.x == 0) {
        bool beg = true;
        for (size_t i = 0; i < tot_bytes; i++) {
            if (beg) {
                printf("\n");
                for (size_t j = 3-(d_skms[i]>>6); j < 3; j++) {
                    printf("%u", (unsigned char)((d_skms[i]>>((2-j)*2)))&0b11);
                } printf(" ");
                beg = false;
            } else {
                for (size_t j = 0; j < (d_skms[i]>>6); j++) {
                    printf("%u", (unsigned char)((d_skms[i]>>((2-j)*2)))&0b11);
                } printf(" ");
                beg = (d_skms[i]>>6)!=3;
            }
        }
    }
}
#endif

__host__ byte* load_SKM_from_file (SKMStoreNoncon &skms_store) {
    byte* d_skms;
    CUDA_CHECK(hipMalloc((void**) &(d_skms), skms_store.tot_size_bytes));
    FILE* fp;
    fp = fopen(skms_store.filename.c_str(), "rb");
    assert(fp);
    byte* tmp;
    tmp = new byte[skms_store.tot_size_bytes];
    assert(fread(tmp, 1, skms_store.tot_size_bytes, fp)==skms_store.tot_size_bytes);
    CUDA_CHECK(hipMemcpy(d_skms, tmp, skms_store.tot_size_bytes, hipMemcpyHostToDevice));
    delete tmp;
    fclose(fp);
    return d_skms;
}

void Extract_Kmers (SKMStoreNoncon &skms_store, T_kvalue k, _out_ T_kmer* &d_kmers, hipStream_t &stream, int BpG=8, int TpB=256) {
    // hipStream_t stream;
    // CUDA_CHECK(hipStreamCreate(&stream));
    
    byte* d_skms;
    
    unsigned long long *d_kmer_store_pos;
    CUDA_CHECK(hipMallocAsync((void**) &(d_kmer_store_pos), sizeof(size_t), stream));
    CUDA_CHECK(hipMemsetAsync(d_kmer_store_pos, 0, sizeof(unsigned long long), stream));

    // ---- copy skm chunks H2D ----
    if (skms_store.to_file) d_skms = load_SKM_from_file(skms_store);
    else {
        CUDA_CHECK(hipMallocAsync((void**) &(d_skms), skms_store.tot_size_bytes, stream));
        int i;
        byte *d_store_pos = d_skms;
        for (i=0; i<skms_store.skm_chunk_bytes.size(); i++) {
            CUDA_CHECK(hipMemcpyAsync(d_store_pos, skms_store.skm_chunks[i], skms_store.skm_chunk_bytes[i], hipMemcpyHostToDevice, stream));
            d_store_pos += skms_store.skm_chunk_bytes[i];
        }
    }
    // cerr<<"debug2"<<endl;
    // CUDA_CHECK(hipStreamSynchronize(stream));
    // ---- GPU work ----
    if (skms_store.tot_size_bytes / 4 <= BpG * TpB) GPU_Extract_Kmers<<<1, skms_store.tot_size_bytes/64+1, 0, stream>>>(d_skms, skms_store.tot_size_bytes, d_kmers, d_kmer_store_pos, k); // 强行debug
    else GPU_Extract_Kmers<<<BpG, TpB, 0, stream>>>(d_skms, skms_store.tot_size_bytes, d_kmers, d_kmer_store_pos, k);
    // GPU_Extract_Kmers_test<<<BpG, TpB, 0, stream>>>(d_skms, skms_store.tot_size_bytes, d_kmers, d_kmer_store_pos, k);
    
    CUDA_CHECK(hipFreeAsync(d_skms, stream));
    CUDA_CHECK(hipFreeAsync(d_kmer_store_pos, stream));
    return;
}

__host__ size_t kmc_counting_GPU_streams (T_kvalue k,
                               vector<SKMStoreNoncon*> skms_stores, CUDAParams &gpars,
                               unsigned short kmer_min_freq, unsigned short kmer_max_freq,
                               _out_ vector<T_kmc> kmc_result_curthread [], int tid,
                               bool GPU_compression = false) {
    // using CUDA Thrust
    int gpuid = (gpars.device_id++)%gpars.n_devices;
    CUDA_CHECK(hipSetDevice(gpuid));
    // V2:
    // if (gpars.gpuid_thread[tid] == -1) {
    //     CUDA_CHECK(hipSetDevice(tid%gpars.n_devices));
    //     gpars.gpuid_thread[tid] = tid%gpars.n_devices;
    // }
    // int gpuid = gpars.gpuid_thread[tid];
    
    size_t return_value = 0;
    int i, n_streams = skms_stores.size();
    hipStream_t streams[n_streams];

    vector<thrust::device_vector<T_kmer>> kmers_d_vec(n_streams); // for 0
    vector<size_t> tot_kmers(n_streams);
    string logs = "GPU "+to_string(gpuid)+":";
    for (i=0; i<n_streams; i++) {
        CUDA_CHECK(hipStreamCreate(&streams[i]));
        // logger->log("GPU "+to_string(gpuid)+" Stream "+to_string(i)+" counting Partition "+to_string(skms_stores[i]->id), Logger::LV_INFO);
        logs += "\tS "+to_string(i)+" Part "+to_string(skms_stores[i]->id)+" "+to_string(skms_stores[i]->tot_size_bytes)+"|"+to_string(skms_stores[i]->kmer_cnt);
        // logger->log(logs, Logger::LV_INFO);
        if (skms_stores[i]->tot_size_bytes != 0) {
            // ---- 0. Extract kmers from SKMStore: ---- 
            kmers_d_vec[i] = thrust::device_vector<T_kmer>(skms_stores[i]->kmer_cnt);
            T_kmer *d_kmers_data = thrust::raw_pointer_cast(kmers_d_vec[i].data());
            // if (GPU_compression) Extract_Kmers_Compressed(*skms_stores[i], k, d_kmers_data, streams[i], gpars.BpG, gpars.TpB, gpuid);
            /*else*/ Extract_Kmers(*skms_stores[i], k, d_kmers_data, streams[i], gpars.BpG, gpars.TpB);
            tot_kmers[i] = kmers_d_vec[i].size();
        }
    }

    thrust::constant_iterator<T_kvalue> ik(k);
    vector<thrust::device_vector<bool>> same_flag_d_vec(n_streams); // for 3
    for (i=0; i<n_streams; i++) {
        if (skms_stores[i]->tot_size_bytes != 0) {
            // CUDA_CHECK(hipStreamSynchronize(streams[i])); // maybe don't need this?
            // ---- 1. convert to canonical kmers ---- 
            thrust::transform(thrust::device.on(streams[i]), kmers_d_vec[i].begin(), kmers_d_vec[i].end(), ik, kmers_d_vec[i].begin(), canonicalkmer());
            // ---- 2. sort: [ABCBBAC] -> [AABBBCC] (kmers_d) ---- 
            thrust::sort(thrust::device.on(streams[i]), kmers_d_vec[i].begin(), kmers_d_vec[i].end()/*, thrust::greater<T_kmer>()*/);
            skms_stores[i]->clear_skm_data(); // only when gpu compression and in-mem
            // ---- 3. find changes: [AABBBCC] -> [0,1,0,1,1,0,1] (same_flag_d) ---- 
            same_flag_d_vec[i] = thrust::device_vector<bool>(kmers_d_vec[i].size());
            thrust::transform(thrust::device.on(streams[i]), kmers_d_vec[i].begin()+1 /*x beg*/, kmers_d_vec[i].end() /*x end*/, kmers_d_vec[i].begin()/*y beg*/, same_flag_d_vec[i].begin()+1/*res beg*/, sameasprev());
        }
    }

    vector<thrust::device_vector<T_read_len>> idx_d_vec(n_streams); // for 4
    vector<thrust::host_vector<T_kmer>> sorted_kmers_h_vec(n_streams); // for 4+
    vector<thrust::host_vector<T_read_len>> idx_h_vec(n_streams); // for 5
    for (i=0; i<n_streams; i++) {
        if (skms_stores[i]->tot_size_bytes != 0) {
            // ---- 3. find changes (cont'd)
            same_flag_d_vec[i][0] = 0; // will it call stream sync?
            // ---- 4. remove same idx: [0123456] [0101101] -> [0,2,5] (idx_d) ----
            idx_d_vec[i] = thrust::device_vector<T_read_len>(kmers_d_vec[i].size());
            thrust::sequence(thrust::device.on(streams[i]), idx_d_vec[i].begin(), idx_d_vec[i].end());
            auto newend_idx_d = thrust::remove_if(thrust::device.on(streams[i]), idx_d_vec[i].begin(), idx_d_vec[i].end(), same_flag_d_vec[i].begin(), thrust::identity<bool>()); // new_end_idx_d is an iterator
            // 4+. copy sorted kmers back to host
            auto newend_sorted_cleared_kmers_d = thrust::remove_if(thrust::device.on(streams[i]), kmers_d_vec[i].begin(), kmers_d_vec[i].end(), same_flag_d_vec[i].begin(), thrust::identity<bool>()); // new_end_idx_d is an iterator
            sorted_kmers_h_vec[i] = thrust::host_vector<T_kmer>(kmers_d_vec[i].begin(), newend_sorted_cleared_kmers_d);
            volatile T_kmer tmp_kmer = sorted_kmers_h_vec[i][0];
            // ---- 5. copy device_vector back to host_vector ----
            idx_h_vec[i] = thrust::host_vector<T_read_len>(idx_d_vec[i].begin(), newend_idx_d);
            idx_h_vec[i].push_back(tot_kmers[i]); // [0,2,5] -> [0,2,5,7] A2 B3 C2
        }
    }
    
    // validation:
    for (i=0; i<n_streams; i++) {
        if (skms_stores[i]->tot_size_bytes == 0) continue;
        size_t total_kmer_cnt = 0;
        T_kmer_cnt cnt;
        for(int j=0; j<idx_h_vec[i].size()-1; j++) {
            cnt = idx_h_vec[i][j+1]-idx_h_vec[i][j] > MAX_KMER_CNT ? MAX_KMER_CNT : idx_h_vec[i][j+1]-idx_h_vec[i][j];
            total_kmer_cnt += idx_h_vec[i][j+1]-idx_h_vec[i][j];
            // Add kmer-cnt to result vector:
            // if (cnt >= kmer_min_freq && cnt <= kmer_max_freq) {
            //     kmc_result_curthread[skms_stores[i]->id].push_back({sorted_kmers_h[idx_h[j]], cnt});
            //     kmc_result_curthread[skms_stores[i]->id].push_back({sorted_kmers_h[j], cnt});
            // }
        }
        assert(total_kmer_cnt == skms_stores[i]->kmer_cnt);
    }
    for (i=0; i<n_streams; i++) {
        if (skms_stores[i]->tot_size_bytes == 0) continue;
        return_value += idx_h_vec[i].size()-1;
    }
    for (i=0; i<n_streams; i++) {
        delete skms_stores[i];//
    }
    logger->log(logs+" "+to_string(return_value), Logger::LV_DEBUG);
    return return_value; // total distinct kmer
}
