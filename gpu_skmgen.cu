#include "hip/hip_runtime.h"
#define _in_
#define _out_

// #define KERNEL_TIME_MEASUREMENT

#define FILTER_KERNEL new_filter2 // modify this to change filter: mm_filter, sign_filter, new_filter, new_filter2
#define STR1(R)  #R
#define STR(R) STR1(R)

#include "gpu_skmgen.h"
#include "types.h"
#include "utilities.hpp"
#include <hip/hip_runtime.h>
#include <hip/hip_runtime_api.h>
// #include "nvcomp/gdeflate.hpp"
// #include "nvcomp.hpp"

#include <vector>
#include <string>
#include <thread>
#include <future>
#include <functional>
#include <iostream>

using namespace std;
// using namespace nvcomp;

__device__ __constant__ static const unsigned char d_basemap[256] = {
    255, 255, 255, 255, 255, 255, 255, 255, // 0..7
    255, 255, 255, 255, 255, 255, 255, 255, // 8..15
    255, 255, 255, 255, 255, 255, 255, 255, // 16..23
    255, 255, 255, 255, 255, 255, 255, 255, // 24..31
    255, 255, 255, 255, 255, 255, 255, 255, // 32..39
    255, 255, 255, 255, 255, 255, 255, 255, // 40..47
    255, 255, 255, 255, 255, 255, 255, 255, // 48..55
    255, 255, 255, 255, 255, 255, 255, 255, // 56..63
    255, 0, 255, 1, 255, 255, 255, 2, // 64..71
    255, 255, 255, 255, 255, 255, 255, 255, // 72..79
    255, 255, 255, 255, 3, 0, 255, 255, // 80..87
    255, 255, 255, 255, 255, 255, 255, 255, // 88..95
    255, 0, 255, 1, 255, 255, 255, 2, // 96..103
    255, 255, 255, 255, 255, 255, 255, 255, // 104..111
    255, 255, 255, 255, 3, 0, 255, 255, // 112..119
    255, 255, 255, 255, 255, 255, 255, 255, // 120..127
    255, 255, 255, 255, 255, 255, 255, 255, // 128..135
    255, 255, 255, 255, 255, 255, 255, 255, // 136..143
    255, 255, 255, 255, 255, 255, 255, 255, // 144..151
    255, 255, 255, 255, 255, 255, 255, 255, // 152..159
    255, 255, 255, 255, 255, 255, 255, 255, // 160..167
    255, 255, 255, 255, 255, 255, 255, 255, // 168..175
    255, 255, 255, 255, 255, 255, 255, 255, // 176..183
    255, 255, 255, 255, 255, 255, 255, 255, // 184..191
    255, 255, 255, 255, 255, 255, 255, 255, // 192..199
    255, 255, 255, 255, 255, 255, 255, 255, // 200..207
    255, 255, 255, 255, 255, 255, 255, 255, // 208..215
    255, 255, 255, 255, 255, 255, 255, 255, // 216..223
    255, 255, 255, 255, 255, 255, 255, 255, // 224..231
    255, 255, 255, 255, 255, 255, 255, 255, // 232..239
    255, 255, 255, 255, 255, 255, 255, 255, // 240..247
    255, 255, 255, 255, 255, 255, 255, 255  // 248..255
};

__device__ __constant__ static const unsigned char d_basemap_compl[256] = { // complement base
    255, 255, 255, 255, 255, 255, 255, 255, // 0..7
    255, 255, 255, 255, 255, 255, 255, 255, // 8..15
    255, 255, 255, 255, 255, 255, 255, 255, // 16..23
    255, 255, 255, 255, 255, 255, 255, 255, // 24..31
    255, 255, 255, 255, 255, 255, 255, 255, // 32..39
    255, 255, 255, 255, 255, 255, 255, 255, // 40..47
    255, 255, 255, 255, 255, 255, 255, 255, // 48..55
    255, 255, 255, 255, 255, 255, 255, 255, // 56..63
    255, 3, 255, 2, 255, 255, 255, 1, // 64..71
    255, 255, 255, 255, 255, 255, 255, 255, // 72..79
    255, 255, 255, 255, 0, 3, 255, 255, // 80..87
    255, 255, 255, 255, 255, 255, 255, 255, // 88..95
    255, 3, 255, 2, 255, 255, 255, 1, // 96..103
    255, 255, 255, 255, 255, 255, 255, 255, // 104..111
    255, 255, 255, 255, 0, 3, 255, 255, // 112..119
    255, 255, 255, 255, 255, 255, 255, 255, // 120..127
    255, 255, 255, 255, 255, 255, 255, 255, // 128..135
    255, 255, 255, 255, 255, 255, 255, 255, // 136..143
    255, 255, 255, 255, 255, 255, 255, 255, // 144..151
    255, 255, 255, 255, 255, 255, 255, 255, // 152..159
    255, 255, 255, 255, 255, 255, 255, 255, // 160..167
    255, 255, 255, 255, 255, 255, 255, 255, // 168..175
    255, 255, 255, 255, 255, 255, 255, 255, // 176..183
    255, 255, 255, 255, 255, 255, 255, 255, // 184..191
    255, 255, 255, 255, 255, 255, 255, 255, // 192..199
    255, 255, 255, 255, 255, 255, 255, 255, // 200..207
    255, 255, 255, 255, 255, 255, 255, 255, // 208..215
    255, 255, 255, 255, 255, 255, 255, 255, // 216..223
    255, 255, 255, 255, 255, 255, 255, 255, // 224..231
    255, 255, 255, 255, 255, 255, 255, 255, // 232..239
    255, 255, 255, 255, 255, 255, 255, 255, // 240..247
    255, 255, 255, 255, 255, 255, 255, 255  // 248..255
};

// raw read is not a significant VRAM usage, no need for 2-bit encoding
// the majority VRAM usage is caused by minimizer (positions) etc...

extern Logger *logger;

// =================================================
// ================ CLASS PinnedCSR ================
// =================================================
    PinnedCSR::PinnedCSR(vector<ReadPtr> &reads) { // for sorting CSR (order the pointers as the sorting result)
        this->n_reads = reads.size();
        size_capacity = 0;
        for (const ReadPtr &read_ptr: reads) {
            size_capacity += read_ptr.len;
        } // about hipHostAlloc https://zhuanlan.zhihu.com/p/188246455
        CUDA_CHECK(hipHostAlloc((void**)(&reads_offs), (this->n_reads+1)*sizeof(T_CSR_cap), hipHostMallocDefault));
        CUDA_CHECK(hipHostAlloc((void**)(&reads_CSR), size_capacity+1, hipHostMallocDefault));
        char *cur_ptr = reads_CSR;
        T_CSR_cap *offs_ptr = reads_offs;
        *offs_ptr = 0;
        for (const ReadPtr &read_ptr: reads) {
            memcpy(cur_ptr, read_ptr.read, read_ptr.len);
            cur_ptr += read_ptr.len;
            offs_ptr++;
            *offs_ptr = *(offs_ptr-1) + read_ptr.len;
        }
    }
    PinnedCSR::~PinnedCSR() {
        CUDA_CHECK(hipHostFree(reads_offs));
        CUDA_CHECK(hipHostFree(reads_CSR));
    }

__global__ void GPU_HPCEncoding (
    _in_ T_read_cnt d_reads_cnt, _out_ T_read_len *d_read_len, 
    _in_ _out_ unsigned char *d_reads, _in_ T_CSR_cap *d_read_offs, 
    bool HPC, _out_ T_read_len *d_hpc_orig_pos = nullptr) {
    int tid = blockDim.x * blockIdx.x + threadIdx.x;
    int n_t = blockDim.x * gridDim.x;
    if (!HPC) { // only calculate read_len in global memory (optional but essential for HPC=true)
        for (T_read_cnt rid = tid; rid < d_reads_cnt; rid += n_t) {
            d_read_len[rid] = d_read_offs[rid+1] - d_read_offs[rid];
        }
        __syncthreads();
        return;
    }
    
    for (T_read_cnt rid = tid; rid < d_reads_cnt; rid += n_t) {
        T_read_len read_len = d_read_offs[rid+1] - d_read_offs[rid];
        T_read_len last_idx = 0, hpc_arr_idx = d_read_offs[tid], j;
        d_hpc_orig_pos[hpc_arr_idx] = 0;
        for (T_read_len i = 1; i < read_len; i++) {
            j = i + d_read_offs[rid];
            last_idx += (i-last_idx) * (d_reads[j] != d_reads[j-1]);
            hpc_arr_idx += (d_reads[j] != d_reads[j-1]);
            d_hpc_orig_pos[hpc_arr_idx] = last_idx;
            d_reads[hpc_arr_idx] = d_reads[j];
        }
        d_read_len[rid] = hpc_arr_idx + 1 - d_read_offs[rid];
    }
    return;
}

// ======== Minimizer Functions ========
// traditional minimizer
__device__ __forceinline__ bool mm_filter(T_minimizer mm, int p) {
    // return mm%101>80; // 20.36
    // return ((mm >> ((p-3)*2)) != 0) /*AAA*/ & (mm >> ((p-3)*2) != 0b000100) /*ACA*/; // 19.94
    // return ((mm >> (p-2)*2) & 0b11) + ((mm >> (p-3)*2) & 0b11) + ((mm >> (p-1)*2) & 0b11); // 20.03
    // return (mm >> (p-3)*2) * ((mm >> (p-5)*2) & 0b111111); // 20.02
    // return ((mm >> ((p-3)*2)) != 0) /*AAA*/ & (mm >> ((p-3)*2) != 0b000100) /*ACA*/ & (mm >> ((p-3)*2) != 0b001000); // 19.92
    // int i=0;
    // int s=0;
    // for (i=1; i<3; i++) {
    //     s += (mm >> ((p-2)*2)) > (mm>>((p-2-i))&0b1111);
    // }
    // return s==0;
    return true;
}
// new design: 2nd/3rd不都为a
__device__ __forceinline__ bool new_filter(T_minimizer mm, int p) {
    return ((mm >> (p-2)*2) & 0b11) + ((mm >> (p-3)*2) & 0b11);
}
__device__ __forceinline__ bool new_filter2(T_minimizer mm, int p) {
    // return ((mm >> ((p-3)*2)) != 0) /*AAA*/ & (mm >> ((p-3)*2) != 0b000100) /*ACA*/; //& (mm >> ((p-3)*2) != 0b001000) /*AGA*/;
    return ((((mm >> ((p-3)*2)) & 0b111011) != 0/*no AAA ACA*/) & ((mm & 0b111111) != 0/*no AAA at last*/));
}
// KMC2 signature
__device__ bool sign_filter(T_minimizer mm, int p) {
    T_minimizer t = mm;
    bool flag = true;
    for (int ii = 0; ii < p-2; ii ++) {
        flag *= ((t & 0b1111) != 0);
        t = t >> 2;
    }
    // printf("%d Minimizer: %x\n", flag & ((mm >> ((p-3)*2)) != 0) /*AAA*/ & (mm >> ((p-3)*2) != 0b000100), mm);
    return flag & (((mm >> ((p-3)*2)) & 0b111011) != 0); /*AAA ACA*/;
}
/*
 * [INPUT]  d_reads in [(Read#0)['A','C','T','G',...], (Read#1)['A','C','T','G',...]]
 * [OUTPUT] d_minimizers in [(Read#0)[mm1, mm?, mm?, ...], (Read#1)...]
 * all thread do one read at the same time with coalesced global memory access
 */
__global__ void GPU_GenMinimizer(
    _in_ T_read_cnt d_reads_cnt, _in_ T_read_len *d_read_len, 
    _in_ unsigned char *d_reads, _in_ T_CSR_cap *d_read_offs, 
    _out_ T_minimizer *d_minimizers, 
    const T_kvalue K_kmer, const T_kvalue P_minimizer) {

    int tid = blockDim.x * blockIdx.x + threadIdx.x;
    int n_t = blockDim.x * gridDim.x;
    int i, j, cur_kmer_i;
    T_minimizer mm_mask = T_MM_MAX >> (sizeof(T_minimizer)*8 - 2*P_minimizer);
    T_minimizer mm_set; // selected minimizer
    T_minimizer mm, new_mm, mm_rc, new_mm_rc; // rc for reverse complement
    
    bool mm_check; // whether is a legal minimizer/signature (filtered by mm_filter)

    for (i=0; i<d_reads_cnt; i++) {
        unsigned char *read = &d_reads[d_read_offs[i]]; // current read
        T_minimizer *minimizer_saving = &(d_minimizers[d_read_offs[i]]);
        T_read_len len = d_read_len[i];
        for (cur_kmer_i=tid; cur_kmer_i <= len-K_kmer; cur_kmer_i+=n_t) { // Coalesced Access
            // gen the first p-mer:
            new_mm = 0;
            for (j = cur_kmer_i; j < cur_kmer_i + P_minimizer; j++) {
                new_mm = (new_mm << 2) | d_basemap[*(read+j)];
            }
            mm_check = FILTER_KERNEL(new_mm, P_minimizer);
            mm = new_mm * mm_check + mm_mask * (!mm_check); // if not a minimizer, let it be maximal (no minimizer can be maximal because canonical)
            
            // gen the first RC p-mer:
            new_mm_rc = 0;
            for (j = cur_kmer_i + P_minimizer - 1; j >= cur_kmer_i; j--) {
                new_mm_rc = (new_mm_rc << 2) | d_basemap_compl[*(read+j)];
            }
            mm_check = FILTER_KERNEL(new_mm_rc, P_minimizer);
            mm_rc = new_mm_rc * mm_check + mm_mask * (!mm_check);

            mm_set = (mm_rc < mm) * mm_rc + (mm_rc >= mm) * mm;////////////
            
            // gen the next p-mers:
            for (j = cur_kmer_i + P_minimizer; j < cur_kmer_i + K_kmer; j++) {
                // gen new minimizers
                new_mm = ((new_mm << 2) | d_basemap[*(read+j)]) & mm_mask;
                new_mm_rc = (new_mm_rc >> 2) | (d_basemap_compl[*(read+j)] << (P_minimizer*2-2));
                // check new minimizers
                mm_check = FILTER_KERNEL(new_mm, P_minimizer);
                mm = new_mm * mm_check + mm * (!mm_check);
                mm_check = FILTER_KERNEL(new_mm_rc, P_minimizer);
                mm_rc = new_mm_rc * mm_check + mm_rc * (!mm_check);
                // set the best minimizer
                mm_set = (mm_set < mm) * mm_set + (mm_set >= mm) * mm;
                mm_set = (mm_set < mm_rc) * mm_set + (mm_set >= mm_rc) * mm_rc;//////////
            }
            minimizer_saving[cur_kmer_i] = mm_set;
        }
    }
    return;
}

__device__ __forceinline__ int _hash_partition (T_minimizer mm, int SKM_partitions) {
    return (~mm) % SKM_partitions;
}
__device__ inline T_skm_len _skm_bytes_required (T_read_len beg, T_read_len end, int k) {
    return sizeof(T_skm_len) + ((beg%4) + end+(k-1)-beg + 3) / 4;
    // return ((beg%3) + end+(k-1)-beg + 3) / 3; // +3 because skm_3x requires an extra empty byte
}
/* [INPUT]  data.minimizers in [[mm1, mm1, mm2, mm3, ...], ...]
 * [OUTPUT] data.superkmer_offs in [[0, 2, 3, ...], ...]
 * [OUTPUT] data.d_skm_part_bytes (size in bytes of each partition)
 * [OUTPUT] data.d_skm_cnt (skm count of each partition)
*/
__global__ void GPU_GenSKMOffs(
    _in_ T_read_cnt d_reads_cnt, _in_ T_read_len *d_read_len, 
    _in_ T_CSR_cap *d_read_offs, 
    _in_ T_minimizer *d_minimizers,
    _out_ T_read_len *d_superkmer_offs,
    _out_ T_skm_partsize *d_skm_part_bytes,
    _out_ T_skm_partsize *d_skm_cnt,
    _out_ T_skm_partsize *d_kmer_cnt,
    const T_kvalue K_kmer, const T_kvalue P_minimizer, const int SKM_partitions) {
        
    int tid = blockDim.x * blockIdx.x + threadIdx.x;
    int n_t = blockDim.x * gridDim.x;

    int p_i;
    bool new_skm;
    T_read_len i;

    // reset shared arrays for skm and kmer counters
    extern __shared__ unsigned int shared_arr[];
    unsigned int *p_skm_cnt = &shared_arr[0];
    unsigned int *p_kmer_cnt = &shared_arr[SKM_partitions];
    for (i=threadIdx.x; i<SKM_partitions; i+=blockDim.x) p_skm_cnt[i] = 0, p_kmer_cnt[i] = 0;
    __syncthreads();
    
    // each thread processes one read at a time
    for (T_read_cnt rid = tid; rid < d_reads_cnt; rid += n_t) {
        T_read_len len = d_read_len[rid];                               // current read length
        T_minimizer *minimizers = &(d_minimizers[d_read_offs[rid]]);    // minimizer list pointer
        T_read_len *skm = &d_superkmer_offs[d_read_offs[rid]];          // superkmer list pointer
        T_read_len last_skm_pos = 0, skm_count = 0;                     // position of the last minimizer; superkmer count
        skm[0] = 0;
        for (i = 1; i <= len-K_kmer; i++) {
            new_skm = (minimizers[i] != minimizers[i-1]/*||i-last_skm_pos+K_kmer >= MAX_SKM_LEN*/);
            skm_count += new_skm; // current minimizer != last minimizer, new skm generated
            last_skm_pos = (!new_skm) * last_skm_pos + (new_skm) * i;
            skm[skm_count] = last_skm_pos; // skm #skm_count (begins from 1) ends at last_skm_pos
            // count skm part sizes
            if (new_skm) {
                p_i = _hash_partition(minimizers[i-1], SKM_partitions);
                atomicAdd(&d_skm_part_bytes[p_i], _skm_bytes_required(skm[skm_count-1], skm[skm_count], K_kmer));
                atomicAdd(&p_skm_cnt[p_i], 1);
                atomicAdd(&p_kmer_cnt[p_i], skm[skm_count] - skm[skm_count-1]);
            }
        }
        // process the last skm
        skm_count += 1;
        skm[skm_count] = len-K_kmer+1;
        p_i = _hash_partition(minimizers[i-1], SKM_partitions);
        atomicAdd(&d_skm_part_bytes[p_i], _skm_bytes_required(skm[skm_count-1], skm[skm_count], K_kmer));
        atomicAdd(&p_skm_cnt[p_i], 1);
        atomicAdd(&p_kmer_cnt[p_i], skm[skm_count] - skm[skm_count-1]);
        
        // set the ending 0 and store skm_count at skm[len-1]
        skm[skm_count+1] = 0;
        skm[len-1] = skm_count;
    }

    __syncthreads();
    for (i=threadIdx.x; i<SKM_partitions; i+=blockDim.x) {
        atomicAdd(&d_skm_cnt[i], p_skm_cnt[i]);
        atomicAdd(&d_kmer_cnt[i], p_kmer_cnt[i]);
    }
    return;
}

__global__ void GPU_ReadCompression(_in_ _out_ unsigned char *d_reads, _in_ T_CSR_cap *d_read_offs, _in_ T_read_len *d_read_len, _in_ T_read_cnt d_reads_cnt) {
    
    unsigned char* cur_read;
    T_read_len len;
    // uchar3 c4; // 3 = BYTE_BASES
    // uchar4 c4;
    u_char tmp;
    T_read_len i, j, last_byte_bases;
    
    // each block process one read:
    for (T_read_cnt i_read = blockIdx.x; i_read < d_reads_cnt; i_read += gridDim.x) {
        len = d_read_len[i_read];
        cur_read = (&(d_reads[d_read_offs[i_read]]));
        // one thread process 4 bases at a time:
        for (i = threadIdx.x * BYTE_BASES; i - threadIdx.x*BYTE_BASES <= len; i += blockDim.x * BYTE_BASES) { // Coalesced Access
            // why "- threadIdx.x" in ending condition? - To ensure each thread will run the same time for __syncthreads().
            if (i + BYTE_BASES <= len) {
                // TO-DO: [experiment] compare with below (check if uchar3 is faster than three single vars)
                // tmp = (d_basemap[cur_read[i]] << 4) | (d_basemap[cur_read[i+1]] << 2) | (d_basemap[cur_read[i+2]]) | 0b11000000;
                // c4 = *(reinterpret_cast<uchar4*>(&cur_read[i])); // load 3 bases at a time
                // c4.x = d_basemap[c4.x]; c4.y = d_basemap[c4.y]; c4.z = d_basemap[c4.z]; c4.w = d_basemap[c4.w]; // convert 4 bases to 2-bit
                // tmp = (c4.x << 6) | (c4.y << 4) | (c4.z << 2) | (c4.w); // generate byte
                tmp = (d_basemap[cur_read[i]] << 6) | (d_basemap[cur_read[i+1]] << 4) | (d_basemap[cur_read[i+2]] << 2) | (d_basemap[cur_read[i+3]]);
            }
            __syncthreads(); // avoid overwriting before raw read base is loaded
            if (i + BYTE_BASES <= len) {
                cur_read[i/BYTE_BASES] = tmp;
            }
        }
        i -= blockDim.x*BYTE_BASES;
        if ((i < len) & (i > len-BYTE_BASES)) { // process the last byte, only 1 thread should be available here
            last_byte_bases = len-i;
            tmp = 0;
            for (j=0; j<last_byte_bases; j++) {
                tmp |= d_basemap[cur_read[i+j]] << (6-j*2);
            }
            cur_read[i/BYTE_BASES] = tmp;
        }
    }
    return;
}

/// @brief Each block process the skms of one read so block size should not be too large.
__global__ void GPU_ExtractSKM (
    _in_ T_read_cnt d_reads_cnt, _in_ T_read_len *d_read_len, _in_ T_CSR_cap *d_read_offs, _in_ unsigned char *d_reads,
    _in_ T_minimizer *d_minimizers,
    _in_ T_read_len *d_skm_offs_inread,
    _in_ T_skm_partsize *d_store_pos, /*_in_ T_skm_partsize *d_skm_cnt, */_out_ u_char *d_skm_store_csr, _in_ T_CSR_cap *d_skmpart_offs, 
    // _in_ T_skm_partsize *d_len_store_pos, _out_ T_skm_len *d_skm_lengths, _in_ T_CSR_cap *d_skmlen_offs, 
    const T_kvalue K_kmer, const T_kvalue P_minimizer, const int SKM_partitions
) {
    T_read_len *cur_read_skm_offs;      // skm offs pointer of current read
    u_char *cur_read;
    T_read_len cur_read_len;            // length in bases
    
    int partition;                      // the partition of the current skm
    T_skm_len skm_size_bytes;           // bytes of current skm
    T_skm_partsize cur_skm_store_pos;   // where to store the current skm (partition's own offs)

    // each block process one read
    for (T_read_cnt rid = blockIdx.x; rid < d_reads_cnt; rid += gridDim.x) {
        cur_read_len = d_read_len[rid];
        cur_read_skm_offs = &(d_skm_offs_inread[d_read_offs[rid]]);
        cur_read = &(d_reads[d_read_offs[rid]]);
        // each thread process one skm
        for (T_read_len i_skm = threadIdx.x+1; i_skm <= cur_read_skm_offs[cur_read_len-1]; i_skm += blockDim.x) { // cur_read_skm_offs[cur_read_len-1]: number of skms of this read
            // printf("%d|%d\n",rid,i_skm);
            // for each skm of the current read, cur_read_skm_offs[0] == 0, loop begins from 1
            // -- store skm --
            partition = _hash_partition (d_minimizers[d_read_offs[rid] + cur_read_skm_offs[i_skm-1]], SKM_partitions);
            skm_size_bytes = _skm_bytes_required(cur_read_skm_offs[i_skm-1], cur_read_skm_offs[i_skm], K_kmer); // beg, end, k
            cur_skm_store_pos = atomicAdd(&d_store_pos[partition], skm_size_bytes); // assign space to store current skm
            d_skm_store_csr[d_skmpart_offs[partition] + cur_skm_store_pos]
             = (T_skm_len)((cur_read_skm_offs[i_skm] - cur_read_skm_offs[i_skm-1] + K_kmer - 1) | ((cur_read_skm_offs[i_skm-1] % BYTE_BASES) << 14));
            memcpy(&d_skm_store_csr[d_skmpart_offs[partition] + cur_skm_store_pos + sizeof(T_skm_len)], &cur_read[cur_read_skm_offs[i_skm-1]/BYTE_BASES], skm_size_bytes);
            if (cur_read_skm_offs[i_skm] - cur_read_skm_offs[i_skm-1] + K_kmer - 1 > 256) printf(" = %d\n", cur_read_skm_offs[i_skm] - cur_read_skm_offs[i_skm-1] + K_kmer - 1);
            assert((cur_read_skm_offs[i_skm] - cur_read_skm_offs[i_skm-1] + K_kmer - 1) < 8192);
            // printf("%u\n", d_skm_lengths[d_skmlen_offs[partition] + cur_skmlen_store_pos] & 0b0011111111111111);
        }
    }
    return;
}

/// @brief Set device CSR offsets begin from 0.
/// @param d_reads_cnt 
/// @param d_read_offs 
/// @param add [0] for setting to zero, [positive] value for adding back
/// @return
__global__ void MoveOffset(_in_ T_read_cnt d_reads_cnt, _in_ _out_ T_CSR_cap *d_read_offs, long long add=0) {
    int tid = blockDim.x * blockIdx.x + threadIdx.x;
    int n_t = blockDim.x * gridDim.x;
    add = add - (add==0) * d_read_offs[0];
    for (T_read_cnt rid = tid; rid <= d_reads_cnt; rid += n_t) {
        d_read_offs[rid] += add;
    }
    return;
}

__host__ size_t GPUReset(int did) {
    cerr<<"now reset GPU "<<did<<endl;
    // do not call it after host malloc
    CUDA_CHECK(hipSetDevice(did));
    CUDA_CHECK(hipDeviceReset());
    // CUDA_CHECK(cudaInitDevice(did, ));
    size_t avail, total;
    hipMemGetInfo(&avail, &total);
    CUDA_CHECK(hipDeviceSynchronize());
    return avail;
}

// provide pinned_reads from the shortest to the longest read
__host__ void GenSuperkmerGPU (PinnedCSR &pinned_reads, 
    const T_kvalue K_kmer, const T_kvalue P_minimizer, bool HPC, CUDAParams &gpars,
    const int SKM_partitions, vector<SKMStoreNoncon*> skm_partition_stores
    ) {
    
    int time_all=0, time_filter=0;

    int gpuid = (gpars.device_id++) % gpars.n_devices;
    CUDA_CHECK(hipSetDevice(gpuid));
    
    hipStream_t streams[gpars.n_streams];
    T_d_data gpu_data[gpars.n_streams];
    T_h_data host_data[gpars.n_streams];
    T_CSR_cap batch_size[gpars.n_streams];      // raw reads size in bytes of the current batch
    T_read_cnt bat_beg_read[gpars.n_streams];

    int i, started_streams;
    for (i=0; i<gpars.n_streams; i++)
        CUDA_CHECK(hipStreamCreate(&streams[i]));
    
    T_read_cnt items_per_stream = gpars.BpG1 * gpars.TpB1 * gpars.items_stream_mul;
    T_read_cnt cur_read = 0, end_read;
    i = 0; // i for which stream
    string logs = "   GPU "+to_string(gpuid)+":";
    while (cur_read < pinned_reads.n_reads) {

        for (i = 0; i < gpars.n_streams && cur_read < pinned_reads.n_reads; i++, cur_read += items_per_stream) {
            // i: which stream

            bat_beg_read[i] = cur_read;
            end_read = min(cur_read + items_per_stream, pinned_reads.n_reads); // the last read in this stream batch
            host_data[i].reads_cnt = gpu_data[i].reads_cnt = end_read-cur_read;
            batch_size[i] = pinned_reads.reads_offs[end_read] - pinned_reads.reads_offs[cur_read]; // read size in bytes
            // logger->log("GPU "+to_string(gpuid)+" Stream "+to_string(i)+":\tread count = "+to_string(gpu_data[i].reads_cnt));
            logs += "\tS "+to_string(i)+"  #Reads "+to_string(gpu_data[i].reads_cnt);

            CUDA_CHECK(hipStreamSynchronize(streams[i]));
            // ---- hipMalloc ----
            // reads (data, offs, len, hpc), minmers, skms (offs, part_byte, cnt)
            // ~ 5000 reads / GB
            CUDA_CHECK(hipMallocAsync((void**) &(gpu_data[i].d_reads), sizeof(u_char) * (batch_size[i]+8), streams[i]));// +8 for uchar4 access overflow // 8192 threads(reads) * 20 KB/read     = 160MB VRAM
            CUDA_CHECK(hipMallocAsync((void**) &(gpu_data[i].d_read_offs), sizeof(T_CSR_cap) * (gpu_data[i].reads_cnt+1), streams[i]));    // 8192 threads(reads) * 8 B/read       =  64MB VRAM
            CUDA_CHECK(hipMallocAsync((void**) &(gpu_data[i].d_read_len), sizeof(T_read_len) * (gpu_data[i].reads_cnt), streams[i]));      // 8192 threads(reads) * 4 B/read       =  32MB VRAM
            if (HPC) {// cost a lot VRAM
                CUDA_CHECK(hipMallocAsync((void**) &(gpu_data[i].d_hpc_orig_pos), sizeof(T_read_len) * (batch_size[i]), streams[i]));      // 8192 threads(reads) * 20K * 4B/read  = 640MB VRAM
            } else {
                gpu_data[i].d_hpc_orig_pos = nullptr;
            }
            CUDA_CHECK(hipMallocAsync((void**) &(gpu_data[i].d_minimizers), sizeof(T_minimizer) * (batch_size[i]), streams[i]));           // 8192 threads(reads) * 20K * 4B/read  = 640MB VRAM
            CUDA_CHECK(hipMallocAsync((void**) &(gpu_data[i].d_superkmer_offs), sizeof(T_read_len) * (batch_size[i]), streams[i]));        // 8192 threads(reads) * 20K * 4B/read  = 640MB VRAM
            CUDA_CHECK(hipMallocAsync((void**) &(gpu_data[i].d_skm_part_bytes), sizeof(T_skm_partsize) * SKM_partitions, streams[i]));
            CUDA_CHECK(hipMemsetAsync(gpu_data[i].d_skm_part_bytes, 0, sizeof(T_skm_partsize) * SKM_partitions, streams[i]));
            CUDA_CHECK(hipMallocAsync((void**) &(gpu_data[i].d_skm_cnt), sizeof(T_skm_partsize) * SKM_partitions, streams[i]));
            CUDA_CHECK(hipMallocAsync((void**) &(gpu_data[i].d_kmer_cnt), sizeof(T_skm_partsize) * SKM_partitions, streams[i]));
            CUDA_CHECK(hipMemsetAsync(gpu_data[i].d_skm_cnt, 0, sizeof(T_skm_partsize) * SKM_partitions, streams[i]));
            CUDA_CHECK(hipMemsetAsync(gpu_data[i].d_kmer_cnt, 0, sizeof(T_skm_partsize) * SKM_partitions, streams[i]));
            
            // ---- copy raw reads to device ----
            CUDA_CHECK(hipMemcpyAsync(gpu_data[i].d_reads, &(pinned_reads.reads_CSR[pinned_reads.reads_offs[cur_read]]), batch_size[i], hipMemcpyHostToDevice, streams[i]));
            CUDA_CHECK(hipMemcpyAsync(gpu_data[i].d_read_offs, &(pinned_reads.reads_offs[cur_read]), sizeof(T_CSR_cap) * (gpu_data[i].reads_cnt+1), hipMemcpyHostToDevice, streams[i]));
            
            // ---- GPU gen skm ----
            #ifdef KERNEL_TIME_MEASUREMENT
            WallClockTimer wct;
            #endif
            MoveOffset<<<gpars.BpG1, gpars.TpB1, 0, streams[i]>>>(
                gpu_data[i].reads_cnt, gpu_data[i].d_read_offs, 0
            );
            #ifdef KERNEL_TIME_MEASUREMENT
            CUDA_CHECK(hipStreamSynchronize(streams[i]));
            #endif
            
            GPU_HPCEncoding<<<gpars.BpG1, gpars.TpB1, 0, streams[i]>>>(
                gpu_data[i].reads_cnt,  gpu_data[i].d_read_len, 
                gpu_data[i].d_reads,    gpu_data[i].d_read_offs, 
                HPC,                    gpu_data[i].d_hpc_orig_pos
            );
            #ifdef KERNEL_TIME_MEASUREMENT
            CUDA_CHECK(hipStreamSynchronize(streams[i]));
            
            WallClockTimer wct2;
            #endif
            GPU_GenMinimizer<<<gpars.BpG1, gpars.TpB1, 0, streams[i]>>>(
                gpu_data[i].reads_cnt,  gpu_data[i].d_read_len,
                gpu_data[i].d_reads,    gpu_data[i].d_read_offs,
                gpu_data[i].d_minimizers, 
                K_kmer, P_minimizer
            );
            #ifdef KERNEL_TIME_MEASUREMENT
            CUDA_CHECK(hipStreamSynchronize(streams[i]));
            time_filter += wct2.stop(true);
            #endif

            GPU_GenSKMOffs<<<gpars.BpG1, gpars.TpB1, 2*SKM_partitions*sizeof(unsigned int), streams[i]>>>(
                gpu_data[i].reads_cnt, gpu_data[i].d_read_len, gpu_data[i].d_read_offs, 
                gpu_data[i].d_minimizers,
                gpu_data[i].d_superkmer_offs,
                gpu_data[i].d_skm_part_bytes,
                gpu_data[i].d_skm_cnt,
                gpu_data[i].d_kmer_cnt,
                K_kmer, P_minimizer, SKM_partitions
            );
            #ifdef KERNEL_TIME_MEASUREMENT
            CUDA_CHECK(hipStreamSynchronize(streams[i]));
            time_all += wct.stop(true);
            #endif
            
            GPU_ReadCompression<<<gpars.BpG1, gpars.TpB1, 0, streams[i]>>>(
                gpu_data[i].d_reads, gpu_data[i].d_read_offs, gpu_data[i].d_read_len, gpu_data[i].reads_cnt
            );
            
            // ---- copy skm partition sizes to host ----
            host_data[i].skm_part_bytes = new T_skm_partsize[SKM_partitions];//1
            host_data[i].skm_cnt = new T_skm_partsize[SKM_partitions];//2
            host_data[i].kmer_cnt = new T_skm_partsize[SKM_partitions];//3
            // CUDA_CHECK(hipStreamSynchronize(streams[i]));
            CUDA_CHECK(hipMemcpyAsync(host_data[i].skm_part_bytes,  gpu_data[i].d_skm_part_bytes,    sizeof(T_skm_partsize) * SKM_partitions, hipMemcpyDeviceToHost, streams[i]));
            CUDA_CHECK(hipMemcpyAsync(host_data[i].skm_cnt,         gpu_data[i].d_skm_cnt,           sizeof(T_skm_partsize) * SKM_partitions, hipMemcpyDeviceToHost, streams[i]));
            CUDA_CHECK(hipMemcpyAsync(host_data[i].kmer_cnt,        gpu_data[i].d_kmer_cnt,          sizeof(T_skm_partsize) * SKM_partitions, hipMemcpyDeviceToHost, streams[i]));
            
            // ---- hipMalloc skm store positions ----
            CUDA_CHECK(hipMallocAsync((void**) &(gpu_data[i].d_store_pos), sizeof(T_skm_partsize) * SKM_partitions, streams[i]));
            CUDA_CHECK(hipMemsetAsync(gpu_data[i].d_store_pos, 0, sizeof(T_skm_partsize) * SKM_partitions, streams[i]));
        }
        started_streams = i;

        // ==== Calc SKM Partition Sizes and Extract SKMs ====
        for (i = 0; i < started_streams; i++) {
            
            CUDA_CHECK(hipStreamSynchronize(streams[i])); // for host skm_part_bytes and skm_cnt
            
            // ---- CPU calc bytes of total skm partition and offsets ----
            host_data[i].tot_skm_bytes = 0;
            host_data[i].skmpart_offs = new T_CSR_cap[SKM_partitions+1];//
            host_data[i].skmpart_offs[0] = 0;
            host_data[i].tot_skm_cnt = 0;
            for (int j = 0; j < SKM_partitions; j++) {
                // assert(host_data[i].skm_part_bytes[j] < 0xffffffffu);
                host_data[i].skmpart_offs[j+1] = host_data[i].skmpart_offs[j] + host_data[i].skm_part_bytes[j];
                host_data[i].tot_skm_bytes += host_data[i].skm_part_bytes[j];
                host_data[i].tot_skm_cnt += host_data[i].skm_cnt[j];
            }
            // ---- hipMalloc skm store ----
            CUDA_CHECK(hipMallocAsync((void**) &(gpu_data[i].d_skm_store_csr), host_data[i].tot_skm_bytes, streams[i]));
            
            // ---- memcpy skm part sizes and offsets to gpu ----
            CUDA_CHECK(hipMallocAsync((void**) &(gpu_data[i].d_skmpart_offs), sizeof(T_CSR_cap) * (SKM_partitions+1), streams[i]));
            CUDA_CHECK(hipMemcpyAsync(gpu_data[i].d_skmpart_offs, host_data[i].skmpart_offs, sizeof(T_CSR_cap) * (SKM_partitions+1), hipMemcpyHostToDevice, streams[i]));

            // ---- GPU extract skms ----
            GPU_ExtractSKM<<<gpars.BpG1, gpars.TpB1, 0, streams[i]>>> (
                gpu_data[i].reads_cnt, gpu_data[i].d_read_len, gpu_data[i].d_read_offs, gpu_data[i].d_reads,
                gpu_data[i].d_minimizers, gpu_data[i].d_superkmer_offs, 
                gpu_data[i].d_store_pos, /*gpu_data[i].d_skm_cnt, */gpu_data[i].d_skm_store_csr, gpu_data[i].d_skmpart_offs,
                K_kmer, P_minimizer, SKM_partitions
            );
            // -- Malloc on host for SKM storage --
            host_data[i].skm_store_csr = new u_char[host_data[i].tot_skm_bytes]; // will not be deleted until program ends
        }

        // ==== Copy SKMs Back to CPU ====
        for (i = 0; i < started_streams; i++) {
            // -- Non-compressed SKM collection (D2H) -- 
            CUDA_CHECK(hipMemcpyAsync(host_data[i].skm_store_csr, gpu_data[i].d_skm_store_csr, host_data[i].tot_skm_bytes, hipMemcpyDeviceToHost, streams[i]));
            
            // TO-DO: add if on task to indicate whether to new and D2H
            if (HPC) {
                host_data[i].hpc_orig_pos = new T_read_len[batch_size[i]];//
                host_data[i].read_len = new T_read_len[gpu_data[i].reads_cnt];//
                CUDA_CHECK(hipMemcpyAsync(host_data[i].hpc_orig_pos, gpu_data[i].d_hpc_orig_pos, sizeof(T_read_len) * batch_size[i], hipMemcpyDeviceToHost, streams[i]));
                CUDA_CHECK(hipMemcpyAsync(host_data[i].read_len, gpu_data[i].d_read_len, sizeof(T_read_len) * host_data[i].reads_cnt, hipMemcpyDeviceToHost, streams[i]));
                // TOxDO: add new reads and new reads_offs
                CUDA_CHECK(hipStreamSynchronize(streams[i]));
                // TOxDO: D2H copy reads and calculate reads_offs
            }
            
            // -- Free device memory --
            if (HPC) CUDA_CHECK(hipFreeAsync(gpu_data[i].d_hpc_orig_pos, streams[i]));
            CUDA_CHECK(hipFreeAsync(gpu_data[i].d_reads, streams[i]));
            CUDA_CHECK(hipFreeAsync(gpu_data[i].d_read_offs, streams[i]));
            CUDA_CHECK(hipFreeAsync(gpu_data[i].d_read_len, streams[i]));
            CUDA_CHECK(hipFreeAsync(gpu_data[i].d_minimizers, streams[i]));
            CUDA_CHECK(hipFreeAsync(gpu_data[i].d_superkmer_offs, streams[i]));
            CUDA_CHECK(hipFreeAsync(gpu_data[i].d_skm_part_bytes, streams[i]));
            CUDA_CHECK(hipFreeAsync(gpu_data[i].d_skm_cnt, streams[i]));
            CUDA_CHECK(hipFreeAsync(gpu_data[i].d_kmer_cnt, streams[i]));
            CUDA_CHECK(hipFreeAsync(gpu_data[i].d_store_pos, streams[i]));
            CUDA_CHECK(hipFreeAsync(gpu_data[i].d_skm_store_csr, streams[i]));
            CUDA_CHECK(hipFreeAsync(gpu_data[i].d_skmpart_offs, streams[i]));
        }
        // ==== CPU Store SKMs ====
        for (i = 0; i < started_streams; i++) {
            CUDA_CHECK(hipStreamSynchronize(streams[i]));
            // process_func(host_data[i]);
            SKMStoreNoncon::save_batch_skms (skm_partition_stores, host_data[i].skm_cnt, host_data[i].kmer_cnt, host_data[i].skmpart_offs, host_data[i].skm_store_csr);
            
            // -- clean host variables --
            if (HPC) {
                delete [] host_data[i].hpc_orig_pos;//
                delete [] host_data[i].read_len;//
            }
            
            delete [] host_data[i].skm_part_bytes;//1
            delete [] host_data[i].skm_cnt;//2
            delete [] host_data[i].kmer_cnt;//3
            delete [] host_data[i].skmpart_offs;//
        }
    }
    logger->log(logs);
    if (time_all!=0)
        logger->log("FILTER: " STR(FILTER_KERNEL) " Kernel Functions Time: ALL = "+to_string(time_all)+"ms FILTER = "+to_string(time_filter)+"ms");
}